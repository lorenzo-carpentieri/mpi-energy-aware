#include "hip/hip_runtime.h"
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <nccl.h>
#include <chrono>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "power-profiler/power_prof.hpp"

#define MAX_RUN 5
#define WARM_UP_RUN 5
#define TIME_TO_ACHIEVE_MS 500
#define POWER_SAMPLING_RATE_MS 5
#define dtype uint8_t
#define MAX_BUF 100
#define BYTE_STEP 8

int main(int argc, char *argv[]) {
    int rank, size;
    std::string log_path;
    if (argc != 2)
        return -1;
    else
        log_path = argv[1];

    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    int numGPUs;
    hipGetDeviceCount(&numGPUs);
    if (numGPUs == 0) {
        std::cerr << "No GPU devices available!" << std::endl;
        MPI_Abort(MPI_COMM_WORLD, 1);
    }

    hipSetDevice(rank % numGPUs);
    ncclComm_t comm;
    ncclUniqueId id;
    if (rank == 0) ncclGetUniqueId(&id);
    MPI_Bcast(&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);
    ncclCommInitRank(&comm, size, id, rank);
    
    size_t *buff_size_byte = (size_t *)malloc(sizeof(size_t) * MAX_BUF);
    size_t num_byte = 1;
    int i = 0;
    for (i = 0; num_byte <= (1ULL << 30); i++) {
        buff_size_byte[i] = num_byte * sizeof(dtype);
        num_byte *= BYTE_STEP;
    }
    const int num_iters = i;

    dtype *d_sendbuf, *d_recvbuf;
    hipMalloc((void **)&d_sendbuf, buff_size_byte[num_iters - 1]);
    hipMalloc((void **)&d_recvbuf, buff_size_byte[num_iters - 1]);

    dtype *h_sendbuf = (dtype *)malloc(buff_size_byte[num_iters - 1]);
    dtype *h_recvbuf = (dtype *)malloc(buff_size_byte[num_iters - 1]);

    hipStream_t stream;
    hipStreamCreate(&stream);

    for (int i = 0; i < WARM_UP_RUN; i++) {
        hipMemcpy(h_sendbuf, d_sendbuf, buff_size_byte[0], hipMemcpyDeviceToHost);
        auto start = std::chrono::high_resolution_clock::now();
        ncclGroupStart();
        for (int r=0; r<size; r++) {
            ncclSend(d_sendbuf, (buff_size_byte[i] / sizeof(dtype)), ncclUint8, r, comm, stream);
            ncclRecv(d_recvbuf, (buff_size_byte[i] / sizeof(dtype)), ncclUint8, r, comm, stream);          
        }
        ncclGroupEnd();

        hipStreamSynchronize(stream);
        auto end = std::chrono::high_resolution_clock::now();
        auto time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
        hipMemcpy(d_recvbuf, h_recvbuf, buff_size_byte[0], hipMemcpyHostToDevice);
    }

    if (rank == 0)
        std::cout << "approach,run,chain_size,byte,mem_cpy_time_ms,time_ms,min_goodput_Gbs" << std::endl;
    hipMemset(d_sendbuf, rank, buff_size_byte[num_iters-1]);
    auto mem_cpy_t_start = std::chrono::high_resolution_clock::now();
    hipMemcpy(h_sendbuf, d_sendbuf, buff_size_byte[num_iters-1], hipMemcpyDeviceToHost);
    auto mem_cpy_t_end = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < num_iters; i++) {
        float avg_time_s = 0;
        int chain_size = 0;

        for (int run = 0; run < MAX_RUN; run++) {
            float ar_time = 0;
            chain_size = 0;

            std::string power_file = log_path + "/a2a_nccl_" + std::to_string(buff_size_byte[i]) + "B"+"_rank"+ std::to_string(rank) + ".pow";
            PowerProfiler powerProf(rank % numGPUs, POWER_SAMPLING_RATE_MS, power_file);
            powerProf.start();
            while (ar_time < (TIME_TO_ACHIEVE_MS * 1000)) {
                auto start_s = std::chrono::high_resolution_clock::now();
                ncclGroupStart();
                for (int r=0; r<size; r++) {
                    ncclSend(d_sendbuf, (buff_size_byte[i] / sizeof(dtype)), ncclUint8, r, comm, stream);
                    ncclRecv(d_recvbuf, (buff_size_byte[i] / sizeof(dtype)), ncclUint8, r, comm, stream);          
                }
                ncclGroupEnd();
                hipStreamSynchronize(stream);
                auto end_s = std::chrono::high_resolution_clock::now();
                ar_time += std::chrono::duration_cast<std::chrono::microseconds>(end_s - start_s).count();
                MPI_Allreduce(MPI_IN_PLACE, &ar_time, 1, MPI_FLOAT, MPI_MAX, MPI_COMM_WORLD);
                chain_size++;
            }
            powerProf.stop();

            float mem_cpy_t = std::chrono::duration_cast<std::chrono::microseconds>(mem_cpy_t_end - mem_cpy_t_start).count();
            
            if (rank == 0) {
                float mem_cpy_t_s = (mem_cpy_t * 2) / 1e+6;
                float data_Gb = static_cast<double>(buff_size_byte[i]) / 1.25e+8;
                float ar_time_s = (ar_time / 1e+6);
                float single_run_time_s = (ar_time_s / chain_size);
                avg_time_s += single_run_time_s;
                std::cout << "a2a_nccl," << "run_" << run << "," << chain_size << "," << buff_size_byte[i] << "," << mem_cpy_t_s * 1000 << "," << single_run_time_s * 1000 << "," << (data_Gb / single_run_time_s) << std::endl;
            }
        }
        if (rank == 0) {
            float data_Gb = static_cast<double>(buff_size_byte[i]) / 1.25e+8;
            avg_time_s /= MAX_RUN;
            std::cout << "a2a_nccl,run_avg," << chain_size << "," << buff_size_byte[i] << ",N/A," << avg_time_s * 1000 << "," << (data_Gb / avg_time_s) << std::endl;
        }
    }
    hipMemcpy(d_recvbuf, h_recvbuf, buff_size_byte[num_iters-1], hipMemcpyHostToDevice);


    ncclCommDestroy(comm);
    hipStreamDestroy(stream);
    hipFree(d_sendbuf);
    hipFree(d_recvbuf);
    free(h_sendbuf);
    free(h_recvbuf);
    MPI_Finalize();

    return 0;
}
