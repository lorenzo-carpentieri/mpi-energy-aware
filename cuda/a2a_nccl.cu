#include "hip/hip_runtime.h"
#include <mpi.h>
#include <hip/hip_runtime.h>
#include "utils/nccl_data_type.hpp"
#include <chrono>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "power-profiler/power_prof.hpp"

#define MAX_RUN 10
#define WARM_UP_RUN 5
#define TIME_TO_ACHIEVE_MS 5000
#define POWER_SAMPLING_RATE_MS 5
#define MAX_BUF 100
#define MESSAGE_SIZE_FACTOR 4

template<typename T>
void run(ncclComm_t& comm,int& rank, int& size, int& numGPUs, std::string& log_path, std::string& csv_path){ 
    ncclDataType_t dtype = nccl_type_traits<T>::type; // define the mapping for T and nccl data type used in collectives

    constexpr size_t ONE_GB = 1024 * 1024 * 1024;
    size_t *buff_size_byte = (size_t *)malloc(sizeof(size_t) * MAX_BUF);
    size_t num_elements=1;

    int i=0;
    while(num_elements * sizeof(T) <= ONE_GB ){
        buff_size_byte[i] = num_elements * sizeof(T);
        num_elements *= MESSAGE_SIZE_FACTOR;
        i++;
    }

    const int num_iters = i;
    T *d_sendbuf, *d_recvbuf;
    hipMalloc((void **)&d_sendbuf, buff_size_byte[num_iters - 1]);
    hipMalloc((void **)&d_recvbuf, buff_size_byte[num_iters - 1]);

    T *h_sendbuf = (T *)malloc(buff_size_byte[num_iters - 1]);
    T *h_recvbuf = (T *)malloc(buff_size_byte[num_iters - 1]);

    hipStream_t stream;
    hipStreamCreate(&stream);

    for (int i = 0; i < WARM_UP_RUN; i++) {
        hipMemcpy(h_sendbuf, d_sendbuf, buff_size_byte[0], hipMemcpyDeviceToHost);
        auto start = std::chrono::high_resolution_clock::now();
        ncclGroupStart();
        for (int r=0; r<size; r++) {
            ncclSend(d_sendbuf, (buff_size_byte[i] / sizeof(T)), dtype, r, comm, stream);
            ncclRecv(d_recvbuf, (buff_size_byte[i] / sizeof(T)), dtype, r, comm, stream);          
        }
        ncclGroupEnd();

        hipStreamSynchronize(stream);
        auto end = std::chrono::high_resolution_clock::now();
        auto time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
        hipMemcpy(d_recvbuf, h_recvbuf, buff_size_byte[0], hipMemcpyHostToDevice);
    }

    std::ofstream csv_file(csv_path, std::ios::app);  // use std::ios::app to append if the file exists
    if (rank == 0){
        csv_file << "approach,run,data_type,chain_size,num_byte,mem_cpy_time_ms,time_ms,min_goodput_Gbs,device_energy,host_energy" << std::endl;
    }


    hipMemset(d_sendbuf, rank, buff_size_byte[num_iters-1]);
    auto mem_cpy_t_start = std::chrono::high_resolution_clock::now();
    hipMemcpy(h_sendbuf, d_sendbuf, buff_size_byte[num_iters-1], hipMemcpyDeviceToHost);
    auto mem_cpy_t_end = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < num_iters; i++) {
        float avg_time_s = 0;
        int chain_size = 0;
        double avg_dev_energy_mj = 0;  // average device energy consumption
        double avg_host_energy_mj = 0; // average host energy consumption
        int host_energy_counter=MAX_RUN;
        for (int run = 0; run < MAX_RUN; run++) {
            float a2a_time = 0;
            chain_size = 0;

            // std::string power_file = log_path + "/a2a_nccl_" + std::to_string(buff_size_byte[i]) + "B"+"_rank"+ std::to_string(rank) + ".pow";
            std::string power_file = log_path + "_" + std::to_string(buff_size_byte[i]) + "B"+"_rank"+ std::to_string(rank) + ".pow";

            PowerProfiler powerProf(rank % numGPUs, POWER_SAMPLING_RATE_MS, power_file);
            powerProf.start();
            while (a2a_time < (TIME_TO_ACHIEVE_MS * 1000)) {
                auto start_s = std::chrono::high_resolution_clock::now();
                ncclGroupStart();
                for (int r=0; r<size; r++) {
                    ncclSend(d_sendbuf, (buff_size_byte[i] / sizeof(dtype)), ncclUint8, r, comm, stream);
                    ncclRecv(d_recvbuf, (buff_size_byte[i] / sizeof(dtype)), ncclUint8, r, comm, stream);          
                }
                ncclGroupEnd();
                hipStreamSynchronize(stream);
                auto end_s = std::chrono::high_resolution_clock::now();
                a2a_time += std::chrono::duration_cast<std::chrono::microseconds>(end_s - start_s).count();
                MPI_Allreduce(MPI_IN_PLACE, &a2a_time, 1, MPI_FLOAT, MPI_MAX, MPI_COMM_WORLD);
                chain_size++;
            }

            double dev_energy_mj = powerProf.stop() / static_cast<double>(chain_size); // mJ for a one collective run
            double host_energy_mj = powerProf.get_host_energy() / static_cast<double>(chain_size); //host energy in mj for one collective run
            
            // Consider the energy consumption consumed by all CPUs and all GPUs of each rank
            MPI_Allreduce(MPI_IN_PLACE, &host_energy_mj, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
            MPI_Allreduce(MPI_IN_PLACE, &dev_energy_mj, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
            // When the enery read by the profiler is negative we skip the value 
            if(host_energy_mj <=0.0){
                host_energy_counter--;
                host_energy_mj=0;
            }
            avg_dev_energy_mj+= dev_energy_mj;
            avg_host_energy_mj+= host_energy_mj;



            float mem_cpy_t = std::chrono::duration_cast<std::chrono::microseconds>(mem_cpy_t_end - mem_cpy_t_start).count();
            
            if (rank == 0) {
                float mem_cpy_t_s = (mem_cpy_t * 2) / 1e+6;
                float data_Gb = static_cast<double>(buff_size_byte[i]) / 1.25e+8;
                float a2a_time_s = (a2a_time / 1e+6);
                float single_run_time_s = (a2a_time_s / chain_size);
                avg_time_s += single_run_time_s;
                csv_file << "a2a_cuda_nccl," << "run_" << run << "," << typeid(T).name() << "," << chain_size << "," << buff_size_byte[i] << "," << mem_cpy_t_s * 1000 << "," << single_run_time_s * 1000 << "," << (data_Gb / single_run_time_s)<< ","<< dev_energy_mj << ","<< host_energy_mj << std::endl;
            }
        }
        if (rank == 0) {
            float data_Gb = static_cast<double>(buff_size_byte[i]) / 1.25e+8;
            avg_time_s /= MAX_RUN;
            avg_dev_energy_mj/=MAX_RUN;
            avg_host_energy_mj/=host_energy_counter;
            csv_file << "a2a_cuda_nccl,run_avg," << typeid(T).name() << "," << chain_size << "," << buff_size_byte[i] << ",N/A," << avg_time_s * 1000 << "," << (data_Gb / avg_time_s) << "," << avg_dev_energy_mj << ","<< avg_host_energy_mj << std::endl;
        }
    }
    hipMemcpy(d_recvbuf, h_recvbuf, buff_size_byte[num_iters-1], hipMemcpyHostToDevice);


    hipStreamDestroy(stream);
    hipFree(d_sendbuf);
    hipFree(d_recvbuf);
    free(h_sendbuf);
    free(h_recvbuf);


}

int main(int argc, char *argv[]) {

    int rank, size;
    std::string log_path;
    std::string csv_path;
    
    if (argc != 3)
        return -1;
    else{   
        log_path = argv[1];
        csv_path = argv[2];
    }


    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    // Detect GPUs for each node
    int numGPUs;
    hipGetDeviceCount(&numGPUs);
        
    if (numGPUs == 0) {
        std::cerr << "No GPU devices available!" << std::endl;
        MPI_Abort(MPI_COMM_WORLD, 1);
    }

    // Local rank for each node
    int local_rank;
    MPI_Comm local_comm;
    MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank, MPI_INFO_NULL, &local_comm);
    MPI_Comm_rank(local_comm, &local_rank);

    
  

    // Bind each local process to a GPU
    hipSetDevice(local_rank % numGPUs);

    ncclComm_t comm;
    ncclUniqueId id;
    if (rank == 0) ncclGetUniqueId(&id);
    MPI_Bcast(&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);
    ncclCommInitRank(&comm, size, id, rank);
    
    // Run with different data type
    // run<uint8_t>(comm, rank, size, numGPUs, log_path, csv_path);
    // run<int>(comm, rank, size, numGPUs, log_path, csv_path);
    run<float>(comm, rank, size, numGPUs, log_path, csv_path);
    // run<double>(comm, rank, size, numGPUs, log_path, csv_path);

    
    ncclCommDestroy(comm);
    MPI_Finalize();

    return 0;
}



