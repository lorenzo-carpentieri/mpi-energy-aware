#include "hip/hip_runtime.h"
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "power-profiler/power_prof.hpp"

#define MAX_RUN 5
#define WARM_UP_RUN 5
#define TIME_TO_ACHIEVE_MS 1000
#define dtype uint8_t
#define MAX_BUF 100
#define BYTE_STEP 8
#define POWER_SAMPLING_RATE_MS 5

int main(int argc, char *argv[]) {
    int rank, size;
    std::string log_path;
    if (argc != 2)
        return -1;
    else
        log_path = argv[1];

    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    int numGPUs;
    hipGetDeviceCount(&numGPUs);
    if (numGPUs == 0) {
        std::cerr << "No GPU devices available!" << std::endl;
        MPI_Abort(MPI_COMM_WORLD, 1);
    }

    hipSetDevice(rank % numGPUs);
    
    size_t *buff_size_byte = (size_t *)malloc(sizeof(size_t) * MAX_BUF);
    size_t num_byte = 1;
    int i = 0;
    for (i = 0; num_byte <= (1ULL << 30); i++) {
        buff_size_byte[i] = num_byte * sizeof(dtype);
        num_byte *= BYTE_STEP;
    }
    const int num_iters = i;

    dtype *d_sendbuf, *d_recvbuf;
    hipMalloc((void **)&d_sendbuf, buff_size_byte[num_iters - 1]);
    hipMalloc((void **)&d_recvbuf, buff_size_byte[num_iters - 1]);

    dtype *h_sendbuf = (dtype *)malloc(buff_size_byte[num_iters - 1]);
    dtype *h_recvbuf = (dtype *)malloc(buff_size_byte[num_iters - 1]);

    for (int i = 0; i < WARM_UP_RUN; i++) {
        hipMemcpy(h_sendbuf, d_sendbuf, buff_size_byte[0], hipMemcpyDeviceToHost);
        auto start = std::chrono::high_resolution_clock::now();
        MPI_Allreduce(h_sendbuf, h_recvbuf, (buff_size_byte[0] / sizeof(dtype)), MPI_UINT8_T, MPI_SUM, MPI_COMM_WORLD);
        auto end = std::chrono::high_resolution_clock::now();
        auto time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
        hipMemcpy(d_recvbuf, h_recvbuf, buff_size_byte[0], hipMemcpyHostToDevice);
    }

    if (rank == 0)
        std::cout << "approach,run,chain_size,byte,mem_cpy_time_ms,time_ms,host_energy_uj,min_goodput_Gbs" << std::endl;
    
    hipMemset(d_sendbuf, rank, buff_size_byte[num_iters-1]);

    for (int i = 0; i < num_iters; i++) {
        double avg_host_energy_uj = 0;
        float avg_time_s = 0;
        float avg_mem_cpy_t_s = 0;
        int chain_size = 0;

        for (int run = 0; run < MAX_RUN; run++) {
            float ar_time = 0;
            double ar_energy_uj = 0;

            std::string power_file = log_path + "/ar_cuda_baseline_" + std::to_string(buff_size_byte[i]) + "B"+"_rank"+ std::to_string(rank) + ".pow";
            PowerProfiler powerProf(rank % numGPUs, POWER_SAMPLING_RATE_MS, power_file);
            powerProf.start();
            chain_size = 0;
            
            auto  mem_cpy_t_start = std::chrono::high_resolution_clock::now();
            hipMemcpy(h_sendbuf, d_sendbuf, buff_size_byte[i], hipMemcpyDeviceToHost);
            auto  mem_cpy_t_end = std::chrono::high_resolution_clock::now();
            
            while (ar_time < (TIME_TO_ACHIEVE_MS * 1000)) {
                auto start_s = std::chrono::high_resolution_clock::now();
                MPI_Allreduce(d_sendbuf, d_recvbuf, (buff_size_byte[i] / sizeof(dtype)), MPI_UINT8_T, MPI_SUM, MPI_COMM_WORLD);
                auto end_s = std::chrono::high_resolution_clock::now();
                ar_time += std::chrono::duration_cast<std::chrono::microseconds>(end_s - start_s).count();
                MPI_Allreduce(MPI_IN_PLACE, &ar_time, 1, MPI_FLOAT, MPI_MAX, MPI_COMM_WORLD);
                chain_size++;
            }

            hipMemcpy(d_recvbuf, h_recvbuf, buff_size_byte[i], hipMemcpyHostToDevice);
            powerProf.stop();
            float mem_cpy_t = std::chrono::duration_cast<std::chrono::microseconds>(mem_cpy_t_end - mem_cpy_t_start).count();
            
            if (rank == 0) {
                float mem_cpy_t_s = (mem_cpy_t*2) / 1e+6;
                float data_Gb = static_cast<double>(buff_size_byte[i]) / 1.25e+8;
                float ar_time_s = (ar_time / 1e+6);
                float single_run_time_s = (ar_time_s / chain_size) + mem_cpy_t_s;
                avg_time_s += single_run_time_s;
                std::cout << "ar_cuda_baseline," << "run_" << run << "," << chain_size << "," << buff_size_byte[i] << "," << mem_cpy_t_s * 1000 << "," << single_run_time_s * 1000 << ",N/A," << (data_Gb / single_run_time_s) << std::endl;
            }
        }
        if (rank == 0) {
            float data_Gb = static_cast<double>(buff_size_byte[i]) / 1.25e+8;
            avg_time_s /= MAX_RUN;
            avg_mem_cpy_t_s/=MAX_RUN;
            std::cout << "ar_cuda_baseline,run_avg," << chain_size << "," << buff_size_byte[i] << ",N/A," << avg_time_s * 1000 << ",N/A," << (data_Gb / avg_time_s) << std::endl;
        }
    }


    hipFree(d_sendbuf);
    hipFree(d_recvbuf);
    free(h_sendbuf);
    free(h_recvbuf);
    MPI_Finalize();

    return 0;
}
